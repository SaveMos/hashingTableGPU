﻿
#include "hip/hip_runtime.h"


#include <iostream>
#include <stdio.h>
#include <vector>
#include <string>

#define NUMBER_OF_CUSTOMERS 1000u // How many struct there are in the vector.

#define MAX_USERNAME_LENGTH 20u
#define MAX_BIO_LENGTH 20u

#define HASH_FUNCTION_SIZE 10u // Size of the output space of the hash function.
#define HASH_SHIFT 6u

#define THREAD_NUMBER 10u // The number of threads you want to use.

#define SAMPLE_FILE_PRINT 1

// USED NAMESPACES
using namespace std;

struct strutturaCustomer {
    char username[MAX_USERNAME_LENGTH]; // Identifier field (must be unique for each customer).
    uint64_t number = 0; // Not unique and expected little field.
    char bio[MAX_BIO_LENGTH]; // Not unique and expected big field.
};

__device__ void gpu_strlen(const char* str, size_t& len) {
    len = 0;
    while (str[len] != '\0') {
        len++;
    }
}

__device__ void bitwise_hash_16(char* str, size_t& size, uint16_t& hash) {
    hash = str[0]; // Il primo valore è il primo carattere della stringa.
    for (uint16_t iter = 1; iter < size; iter++) {
        hash += (hash << HASH_SHIFT) + str[iter];
        // Hash bitwise: shift a sinistra di un certo numero di posizioni e poi aggiungi il carattere corrente
    }
    hash %= HASH_FUNCTION_SIZE; // Il digest deve essere all'interno dell'intervallo di output della funzione hash.
}

__global__ void processCustomers(strutturaCustomer* customers, uint64_t size , strutturaCustomer** res, float* overflowIndexes) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    uint16_t hash;
    size_t len = 0;

    // Ogni thread elabora un subset di elementi nell'array customers
    while (idx < size) {
        gpu_strlen(customers[idx].username, len);
        bitwise_hash_16(customers[idx].username, len, hash);

        int index = atomicAdd(&overflowIndexes[hash], 1);
        res[hash][index] = customers[idx];

        idx += blockDim.x * gridDim.x;
    }
    __syncthreads();
}

void cudaMemoryInfo(){
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    std::cout << "Memoria libera sulla GPU: " << freeMem / (1024*1024)<< "/" << totalMem / (1024*1024) << std::endl;
}


int main() {
    cout << "prova partenza" << endl;
    
    uint64_t i = 0, j = 0;
    string username = "";
    strutturaCustomer h_customers[NUMBER_OF_CUSTOMERS]; // Array delle strutture dati sulla CPU
    strutturaCustomer h_res[HASH_FUNCTION_SIZE][NUMBER_OF_CUSTOMERS];
    float h_overflowIndexes[HASH_FUNCTION_SIZE]; 
   
    // Inizializzazione dei dati dei clienti (esempio)
    for (i = 0; i < NUMBER_OF_CUSTOMERS; ++i) {
        username = "user_" + std::to_string(i);
        strcpy(h_customers[i].username, username.c_str());
        h_customers[i].number = i;
        username = "Bio for user_" + std::to_string(i);
        strcpy(h_customers[i].bio, username.c_str());
    }
 
    // Inizializzazione degli indici di overflow.
    for (i = 0; i < HASH_FUNCTION_SIZE; i++) {
        h_overflowIndexes[i] = 0.0f;
    }


    float* d_overflowIndexes;
    hipMalloc((void**)&d_overflowIndexes, NUMBER_OF_CUSTOMERS * sizeof(float)); // Allocazione della memoria sulla GPU per h_overflowIndexes
    hipMemcpy(d_overflowIndexes, h_overflowIndexes, NUMBER_OF_CUSTOMERS * sizeof(float), hipMemcpyHostToDevice);   // Copia dei dati dalla CPU alla GPU per h_overflowIndexes

    strutturaCustomer* d_customers;
    hipMalloc((void**)&d_customers, NUMBER_OF_CUSTOMERS * sizeof(strutturaCustomer));  // Allocazione della memoria sulla GPU per h_customersv
    hipMemcpy(d_customers, h_customers, NUMBER_OF_CUSTOMERS * sizeof(strutturaCustomer), hipMemcpyHostToDevice); // Copia dei dati dalla CPU alla GPU per h_customers

   
    strutturaCustomer** d_res;  // Creiamo la tabella di hash nella GPU
    hipMalloc((void**)&d_res, HASH_FUNCTION_SIZE * sizeof(strutturaCustomer*));

    for(i = 0 ; i < HASH_FUNCTION_SIZE; i++){
        strutturaCustomer* row;
        hipMalloc((void**)&row, NUMBER_OF_CUSTOMERS * sizeof(strutturaCustomer));
        hipMemcpy(d_res+i, &row, sizeof(strutturaCustomer*), hipMemcpyHostToDevice); // Copia dei dati dalla CPU alla GPU per h_customers
    }

    processCustomers<<<NUMBER_OF_CUSTOMERS / THREAD_NUMBER , THREAD_NUMBER >>>(d_customers , NUMBER_OF_CUSTOMERS , d_res , d_overflowIndexes);

    hipDeviceSynchronize();  // Sincronizza la GPU per assicurarsi che il kernel sia stato completato.
    
     // Copia dei risultati dalla GPU alla CPU
    for(i = 0; i < HASH_FUNCTION_SIZE; i++){
        strutturaCustomer* row;
        hipMemcpy(&row, d_res + i, sizeof(strutturaCustomer*), hipMemcpyDeviceToHost);
        hipMemcpy(h_res[i], row, NUMBER_OF_CUSTOMERS * sizeof(strutturaCustomer) , hipMemcpyDeviceToHost);
    }
   
   /*
    for(i = 0; i < HASH_FUNCTION_SIZE; i++){
        if(strlen(h_res[i][0].username) == 0){
            continue;
        }
        cout << i << ") ";
        for(j = 0; j < NUMBER_OF_CUSTOMERS ; j++){
            if(strlen(h_res[i][j].username) == 0){
                break;
            }
            cout << h_res[i][j].username << " -> ";
        }
        cout << endl;
    }
    */
   

    cout << "Inizio deallocazione" << endl;

    // DEALLOCAZIONE

    hipFree(d_customers);   // Deallocazione della memoria sulla GPU per h_customers

    // Deallocazione della memoria sulla GPU per h_res
    for (i = 0; i < HASH_FUNCTION_SIZE; i++) {
        hipFree(d_res[i]);
    }
    hipFree(d_res);

    return 0;
}
