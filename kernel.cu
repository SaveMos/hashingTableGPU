#include "hip/hip_runtime.h"
﻿// CUDA libraries
#include "hip/hip_runtime.h"
#include ""

// Used libraries
#include <stdio.h>
#include <vector>
#include <chrono> // Library for the steady clock.
#include <thread>

#include "utilityFile.h"

// Data structure array configuration
#define NUMBER_OF_CUSTOMERS 1000u // How many struct there are in the vector.

#define THREAD_NUMBER 32u

// Hash configuration
#define HASH_FUNCTION_SIZE 1027u // Size of the output space of the hash function.

// Used namespaces
using namespace std;

// The target data structure.
struct strutturaCustomer
{
    string username;     // Identifier field (must be unique for each customer).
    uint64_t number = 0; // Not unique and expected little field.
    string bio;          // Not unique and expected big field.
};

__device__ void gpu_strlen(char *str, size_t &len)
{
    len = 0;
    while (str[len] != '\0')
    {
        len++;
    }
}

// GPU function for compute the 16-bit hash of a string.
__device__ void bitwise_hash_16(char *str, size_t size, uint16_t &hash)
{
    hash = str[0];
    for (size_t iter = 1; iter < size; iter++)
    {
        hash = (hash << HASH_SHIFT) + str[iter];
    }
    hash %= HASH_FUNCTION_SIZE; // Il digest deve essere all'interno dell'intervallo di output della funzione hash.
}
/*
__global__ void processCustomers(char **customers, uint64_t size, uint16_t *hashes)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t len = 0;
    uint16_t hash;
    // Ogni thread elabora un subset di elementi nell'array customers
    while (idx < size)
    {
        gpu_strlen(customers[idx], len);
        bitwise_hash_16(customers[idx], len, hash);
        hashes[idx] = hash;
        idx += blockDim.x * gridDim.x;
    }
}*/


__global__ void processCustomers(char **customers, uint64_t size, uint16_t *hashes)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t len = 0;
    uint16_t hash;

    if (idx < size)
    {
        // Calculate the length of the string
        gpu_strlen(customers[idx], len);

        // Calculate the hash
        bitwise_hash_16(customers[idx], len, hash);

        // Store the hash in the output array
        //printf("ahaha %d\n" , hash);
        hashes[idx] = hash;
    }
}


// Macro per controllare eventuali errori nella GPU.
#define CUDA_CHECK_RETURN(value)                                          \
    {                                                                     \
        hipError_t _m_cudaStat = value;                                  \
        if (_m_cudaStat != hipSuccess)                                   \
        {                                                                 \
            fprintf(stderr, "Error %s at line %d in file %s\n",           \
                    hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
            exit(1);                                                      \
        }                                                                 \
    }

// Macro per far funzionare le "<<<>>>"
#ifndef __INTELLISENSE__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

void threadCodeHashing(vector<strutturaCustomer> &customers, char **h_customers, uint8_t id)
{
    strutturaCustomer c = {"a", 0, "Insert his bio"}; // Temporary Customer structure, recycled some times in the code to increase the chance of cache hit.
    string username;                                  // Temporary variable, recycled some times in the code to increase the chance of cache hit
    uint64_t target = id;
    const uint64_t size = customers.size();

    while (target < size)
    {
        c.username = "user_" + to_string(target);
        customers.at(target) = c; // Insert the user in the list.

        h_customers[target] = new char[c.username.length() + 1]; // Aggiunto 1 per il terminatore null.
        strcpy(h_customers[target], c.username.c_str());

        target += THREAD_NUMBER_CPU;
    }
}

void threadCodeBuildTable(vector<strutturaCustomer> &customers, uint16_t *hashes, vector<vector<strutturaCustomer>> &ret, uint8_t id)
{
    uint64_t target = id;
    const uint64_t size = customers.size();
    while (target < size)
    {
        ret.at((hashes[target] % HASH_FUNCTION_SIZE)).push_back(customers[target]);
        target += THREAD_NUMBER_CPU;
    }
}

int main()
{
    uint64_t i = 0, count = 0;
    vector<strutturaCustomer> customers(NUMBER_OF_CUSTOMERS);  // The list of the customers.
    vector<vector<strutturaCustomer>> ret(HASH_FUNCTION_SIZE); // The final hashing table.

    uint16_t *hashes = new uint16_t[NUMBER_OF_CUSTOMERS];
    char **h_customers = new char *[NUMBER_OF_CUSTOMERS];

    hipEvent_t tic, toc; // Variables for compute the elapsed time.
    float elapsed = 0.0f; // Variable for compute the elapsed time.

    vector<thread> threadMixer(THREAD_NUMBER_CPU - 1); // Vector of the threads descriptors.
    uint8_t ithread;                                   // Iterator variable.

    decltype(std::chrono::steady_clock::now()) start_steady, end_steady; // The definition of the used timer variables.

    (hipEventCreate(&tic));
    (hipEventCreate(&toc));

    // Inizializzazione dei dati dei clienti (esempio)

    for (ithread = 0; ithread < THREAD_NUMBER_CPU - 1; ithread++)
    { // For each started thread...
        thread thread_i(
            threadCodeHashing, // The thread function.
            ref(customers),    // The customers array.
            ref(h_customers),
            ithread // The thread's id.
        );
        threadMixer.at(ithread) = move(thread_i); // Add the thread descriptor to the thread descriptor vector.
    }

    // The main thread too contribute to the generation of the data stucture.
    threadCodeHashing(
        ref(customers), // The thread function.
        ref(h_customers),
        THREAD_NUMBER_CPU - 1 // The thread's id.
    );

    // Now the father wait for all the started threads to finish their execution.
    for (ithread = 0; ithread < THREAD_NUMBER_CPU - 1; ithread++)
    {
        threadMixer[ithread].join(); // Join the i� thread.
    }

    start_steady = std::chrono::steady_clock::now(); // Start measuring the execution time of the main process.

    //cout << "Inizializzazione delle strutture dati..." << endl;
    // Allocazione overflow indexes in GPU.
    uint16_t *d_hashes = 0;
    (hipMalloc((void **)&d_hashes, NUMBER_OF_CUSTOMERS * sizeof(uint16_t))); // Allocazione della memoria sulla GPU per h_overflowIndexes

    //cout << "Vettore hashes generato e allocato in GPU!" << endl;
    // Allocazione customers in GPU.
    char **d_customers; // Creiamo la tabella di hash nella GPU
    uint8_t size_str;
    char *d_username;
    (hipMalloc((void **)&d_customers, NUMBER_OF_CUSTOMERS * sizeof(char *)));
    for (i = 0; i < NUMBER_OF_CUSTOMERS; i++)
    {
        size_str = customers[i].username.length() + 1;
        (hipMalloc((void **)&d_username, size_str * sizeof(char)));                               // Copia del nome utente dalla CPU alla GPU
        (hipMemcpy(d_username, h_customers[i], size_str * sizeof(char), hipMemcpyHostToDevice)); // Aggiornamento del puntatore del nome utente nella struttura dati sul device
        (hipMemcpy(&(d_customers[i]), &d_username, sizeof(char *), hipMemcpyHostToDevice));
    }

    /*
    cout << "Vettore customers generato e allocato in GPU!" << endl;
    cout << endl;
    cout << endl;

    cout << "Inizio del nucleo." << endl;
    */

    (hipEventRecord(tic, 0));
    processCustomers<<<NUMBER_OF_CUSTOMERS / THREAD_NUMBER, THREAD_NUMBER>>>(d_customers, NUMBER_OF_CUSTOMERS, d_hashes);
    // processCustomers KERNEL_ARGS2(NUMBER_OF_CUSTOMERS / THREAD_NUMBER, THREAD_NUMBER) (d_customers, NUMBER_OF_CUSTOMERS, d_hashes);

    (hipEventRecord(toc, 0));

    (hipDeviceSynchronize()); // Sincronizza la GPU per assicurarsi che il kernel sia stato completato.

    (hipEventSynchronize(toc)); // synchronize the event

    (hipEventElapsedTime(&elapsed, tic, toc)); // Compute the elapsed time

    /*
    cout << "Fine del nucleo." << endl;
    cout << endl;
    cout << endl;

    cout << "Copia dei risultati..." << endl;
    */

  

    (hipMemcpy(hashes, d_hashes, NUMBER_OF_CUSTOMERS * sizeof(uint16_t), hipMemcpyDeviceToHost)); // Copia dei risultati dalla GPU alla CPU.
    cout << "Risultati copiati in memoria!" << endl;

    // Costruzione della tabella di hashing.
    for (ithread = 0; ithread < THREAD_NUMBER_CPU - 1; ithread++)
    { // For each started thread...
        thread thread_i(
            threadCodeBuildTable, // The thread function.
            ref(customers),       // The customers array.
            ref(hashes),
            ref(ret),
            ithread // The thread's id.
        );
        threadMixer.at(ithread) = move(thread_i); // Add the thread descriptor to the thread descriptor vector.
    }

    // The main thread too contribute to the generation of the data stucture.
    threadCodeBuildTable(
        ref(customers), // The thread function.
        ref(hashes),
        ref(ret),
        THREAD_NUMBER_CPU - 1 // The thread's id.
    );

    // Now the father wait for all the started threads to finish their execution.
    for (ithread = 0; ithread < THREAD_NUMBER_CPU - 1; ithread++)
    {
        threadMixer[ithread].join(); // Join the i� thread.
    }

    end_steady = std::chrono::steady_clock::now();                                      // Measure the execution time of the main process when all the threads are ended.
    std::chrono::duration<double> elapsed_seconds_high_res = end_steady - start_steady; // Compute the execution time.
    const double time = elapsed_seconds_high_res.count();                               // Return the total execution time.

    for (i = 0; i < HASH_FUNCTION_SIZE; i++)
    {
        count += ret[i].size();
    }

    if (count == NUMBER_OF_CUSTOMERS)
    {
        cout << "Tabella di hash costruita con successo!" << endl;
    }
    else
    {
        cout << "Tabella di hash non costruita, errore!" << endl;
    }

    cout << endl;
    cout << endl;

    cout << "Inizio deallocazione..." << endl;

    // DEALLOCAZIONE
    for (i = 0; i < NUMBER_OF_CUSTOMERS; ++i)
    {
        char *d_username;
        (hipMemcpy(&d_username, &d_customers[i], sizeof(char *), hipMemcpyDeviceToHost));
        (hipFree(d_username));
    }

    (hipFree(d_customers)); // Deallocazione della memoria sulla GPU per d_customers.
    (hipFree(d_hashes));    // Deallocazione della memoria sulla GPU per d_hashes.
    // cout << "Deallocazione GPU completata!" << endl;

    // Rilascio della memoria CPU allocata
    for (i = 0; i < NUMBER_OF_CUSTOMERS; ++i)
    {
        delete[] h_customers[i];
    }
    delete[] h_customers;
    delete[] hashes;
    customers.clear(); // Polizia del vettore originale.

    // cout << "Deallocazione CPU completata!" << endl;

    // Free the two events tic and toc
    (hipEventDestroy(tic));
    (hipEventDestroy(toc));
    // cout << "Deallocazione Eventi Timer completata!" << endl;

    if (PRINT_CHECKS)
    {
        cout << "-----------------------------------------" << endl;
        cout << "Tempo di esecuzione del nucleo: " << elapsed << " ms" << endl;
        cout << "Tempo di esecuzione totale : " << time << " s" << endl;
        cout << "-----------------------------------------" << endl;
    }

    if (SAMPLE_FILE_PRINT)
    {
        // elapsed must be float, but the function wants double
        printToFile(static_cast<double>(elapsed), "kernel.csv"); // Print the sample in the '.csv' file.
        insertNewLine("kernel.csv");
        printToFile(time, "total.csv"); // Print the sample in the '.csv' file.
        insertNewLine("total.csv");
    }

    return 0;
}
